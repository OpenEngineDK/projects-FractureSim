#include "hip/hip_runtime.h"
#include "CUDA.h"

#include "Body.h"


Body::Body() {}

Body::Body(unsigned int size) {
    CHECK_FOR_CUDA_ERROR();
    numTetrahedra = size;
    
    tetrahedra = 
        (Tetrahedron*) malloc(numTetrahedra*sizeof(Tetrahedron));

    hipMalloc((void**)&(shape_function_deriv), 
               sizeof(ShapeFunctionDerivatives) * numTetrahedra);

    writeIndices = (int4*)malloc(sizeof(int4) * size);
    volume = (float*)malloc(sizeof(float) * size);
    CHECK_FOR_CUDA_ERROR();
}

void Body::ConvertToCuda() {
    CHECK_FOR_CUDA_ERROR();
        Tetrahedron *dTets;
        hipMalloc((void**)&dTets, sizeof(Tetrahedron)*numTetrahedra);
        CHECK_FOR_CUDA_ERROR();
        hipMemcpy(dTets, tetrahedra,
                   sizeof(Tetrahedron)*numTetrahedra , hipMemcpyHostToDevice); 
        CHECK_FOR_CUDA_ERROR();
        free(tetrahedra);
        this->tetrahedra = dTets;

        float* dVolume;
        hipMalloc((void**)&dVolume,
                   sizeof(float) * numTetrahedra);
        CHECK_FOR_CUDA_ERROR();
        hipMemcpy(dVolume, volume,
                   sizeof(float) * numTetrahedra, hipMemcpyHostToDevice);
        CHECK_FOR_CUDA_ERROR();
        free(volume);
        this->volume = dVolume;

        int4* dWriteIndices;
        hipMalloc((void**)&(dWriteIndices),
                   sizeof(int4) * numWriteIndices);
        CHECK_FOR_CUDA_ERROR();
        hipMemcpy(dWriteIndices, writeIndices, 
                   sizeof(int4) * numWriteIndices,
                   hipMemcpyHostToDevice);
        free(writeIndices);
        writeIndices = dWriteIndices;
        CHECK_FOR_CUDA_ERROR();
}

void Body::DeAlloc() {
    CHECK_FOR_CUDA_ERROR();
    hipFree(tetrahedra);
    hipFree(shape_function_deriv);
    hipFree(writeIndices);
    hipFree(volume);
    CHECK_FOR_CUDA_ERROR();
}

void Body::Print() {
    for (unsigned int i=0; i<numTetrahedra; i++) {
        Tetrahedron id = tetrahedra[i];
        printf("b[%i] = (%i,%i,%i,%i)\n", i, id.x, id.y, id.z, id.w);
    }
}

