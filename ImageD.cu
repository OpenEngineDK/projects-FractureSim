#include "hip/hip_runtime.h"
#ifndef __IMAGED_CU__
#define __IMAGED_CU__

//---------------------------------------------------------------------------

#include "ImageD.h"

//---------------------------------------------------------------------------

//inline 

__host__ float *newCUDAImageF(ImageD **maskd, ImageD *maskh, bool copy_data = true)
{
  float * datapos;
//  printf(hipGetErrorString(hipMalloc((void **)&datapos, maskh->dim.x*maskh->dim.y*maskh->dim.z*sizeof(float))));
//  printf(hipGetErrorString(hipMalloc((void **)maskd, sizeof(ImageD))));
//  printf("newCUDAImageF: %i \n",maskh->dim.x);

  CUDA_SAFE_CALL(hipMalloc((void **)&datapos, maskh->dim.x*maskh->dim.y*maskh->dim.z*sizeof(float)));
  CUDA_SAFE_CALL(hipMalloc((void **)maskd, sizeof(ImageD)));

  //hipGetErrorString(hipMalloc((void **)&datapos, maskh->dim.x*maskh->dim.y*maskh->dim.z*sizeof(float)));
  //hipGetErrorString(hipMalloc((void **)maskd, sizeof(ImageD)));

 // copy image data+dimension
  CUDA_SAFE_CALL(hipMemcpy(*maskd, maskh, sizeof(ImageD), hipMemcpyHostToDevice)); 

 CUDA_SAFE_CALL(hipMemcpy(*maskd, &datapos, sizeof(float*), hipMemcpyHostToDevice)); 
// printf("datapos: %i\n", datapos);
// assert(datapos != 0);
// assert(datapos == (*maskd)->data);

// printf("newCUDAImageF: %i == %i \n",datapos,(*maskd)->data);
 
  // copy data
  if (copy_data)
    CUDA_SAFE_CALL(hipMemcpy(datapos, maskh->data, maskh->dim.x*maskh->dim.y*maskh->dim.z*sizeof(float), hipMemcpyHostToDevice)); 
// printf("newCUDAImageF: %i \n",maskh->dim.x);
// printf("newCUDAImageF: %i \n",(*maskd)->dim.x);

  return datapos;
}



//---------------------------------------------------------------------------

ImageD * newImage(int3 dim, float3 scale, float3 origin, bool allocate_space)
{
  ImageD *res = (ImageD *)malloc(sizeof(ImageD));
  res->dim = dim;
  res->origin = origin;
  res->scale = scale;

  if (allocate_space)
  {
	 res->data = (float*) malloc(res->dim.x*res->dim.y*res->dim.z*sizeof(float));

	 for(int i=0;i<res->dim.x*res->dim.y*res->dim.z;i++) res->data[i]=0;
  }
  return res;
}

//---------------------------------------------------------------------------

void freeCUDAImage(ImageD* img)
{
	CUDA_SAFE_CALL(hipFree(img->data));
	CUDA_SAFE_CALL(hipFree(img));
}

//---------------------------------------------------------------------------

void freeImage(ImageD* img)
{
	free(img->data);
	free(img);
}

//---------------------------------------------------------------------------

#endif /* __IMAGED_CU__ */

