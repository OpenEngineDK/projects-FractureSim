
#include "CudaMem.h"


hipError_t CudaMemAlloc(void** devPtr, size_t count) {
    return hipMalloc(devPtr, count);
}

hipError_t CudaMemset(void* devPtr, int value, size_t count) {
    return hipMemset(devPtr, value, count);
}

hipError_t CudaMemcpy( void* dst, const void* src, size_t count, enum hipMemcpyKind kind ) {
    return hipMemcpy(dst, src, count, kind);
} 

hipError_t CudaFree(void* devPtr) {
    return hipFree(devPtr);
}
