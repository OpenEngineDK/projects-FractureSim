#include "CudaMem.h"
#include "CUDA.h"

#include <map>

static size_t alloced = 0;
static std::map<void*, unsigned int> memMap;
static size_t glAlloced = 0;
static std::map<unsigned int, unsigned int> glMemMap;

hipError_t CudaMemAlloc(void** devPtr, size_t count) {
    //printf("cuda alloced memory: %lu bytes\n", count);

    alloced += count;
    hipError_t errorCode = hipMalloc(devPtr, count);
    memMap[*devPtr] = count;
    return errorCode;
}

hipError_t CudaMemset(void* devPtr, int value, size_t count) {
    return hipMemset(devPtr, value, count);
}

hipError_t CudaMemcpy( void* dst, const void* src, size_t count, enum hipMemcpyKind kind ) {
    return hipMemcpy(dst, src, count, kind);
} 

hipError_t CudaFree(void* devPtr) {
    std::map<void*, unsigned int>::iterator iter = memMap.find(devPtr);
    if (iter == memMap.end()) {
        printf("dealloc of unalloced memory, with pointer: %lu\n",
               (unsigned long) devPtr);
        exit(-1);
    }
    alloced -= memMap[devPtr];
    return hipFree(devPtr);
}

unsigned int AllocGLBuffer(unsigned int byteSize) {
    // create buffer object
    GLuint vboID = 0;
    glGenBuffers( 1, &vboID);
    // TODO: error check genBuffer
    //printf("glGenBufferID: %u\n", vboID);
    // Bind buffer
    glBindBuffer( GL_ARRAY_BUFFER, vboID);
    // initialize buffer object
    glBufferData( GL_ARRAY_BUFFER, byteSize, NULL, GL_DYNAMIC_DRAW);
    // Unbind buffer
    glBindBuffer( GL_ARRAY_BUFFER, 0);
    // Register buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLRegisterBufferObject(vboID));
    // Check for errors
    CHECK_FOR_GL_ERROR();

    //printf("gl alloced memory: %u bytes\n", byteSize);

    glAlloced += byteSize;
    glMemMap[vboID] = byteSize;
    return vboID;
}

void FreeGLBuffer(unsigned int id) {
    std::map<unsigned int, unsigned int>::iterator iter = glMemMap.find(id);
    if (iter == glMemMap.end()) {
        printf("error in dealloc of gl unalloced memory, with id %u\n", id);
        exit(-1);
    }
    glAlloced -= glMemMap[id];
    glDeleteBuffers(1, &id);
    CHECK_FOR_GL_ERROR();
}

void PrintAllocedMemory() {
    //printf("cuda alloced memory: %lu bytes\n", alloced);
    //printf("cuda alloced memory: %f kb\n", alloced/1024.0f);
    printf("cuda alloced memory: %f Mb\n", alloced/1024.0f/1024.0f);
    //printf("gl alloced memory: %lu bytes\n", glAlloced);
    //printf("gl alloced memory: %f kb\n", glAlloced/1024.0f);
    printf("gl alloced memory: %f Mb\n", glAlloced/1024.0f/1024.0f);
}
