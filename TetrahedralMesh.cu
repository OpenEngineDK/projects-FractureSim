#include "hip/hip_runtime.h"
#include "TetrahedralMesh.h"
#include "CUDA.h"

VertexPool::VertexPool(unsigned int size) {
    CHECK_FOR_CUDA_ERROR();
    this->size = size;
    data = (Point*) malloc(size*sizeof(Point));

    mass = (float*)malloc(sizeof(float) * size);
    memset(mass, 0, sizeof(float) * size);

	hipMalloc((void**)&(ABC), sizeof(float4) * size);
	hipMalloc((void**)&(Ui_t), sizeof(float4) * size);
	hipMalloc((void**)&(Ui_tminusdt), sizeof(float4) * size);
	hipMalloc((void**)&(externalForces), sizeof(float4) * size);
    CHECK_FOR_CUDA_ERROR();

	hipMemset(externalForces, 0, sizeof(float4) * size);
	hipMemset(Ui_t, 0, sizeof(float4) * size);
	hipMemset(Ui_tminusdt, 0, sizeof(float4) * size);
    CHECK_FOR_CUDA_ERROR();

}

void VertexPool::ConvertToCuda() {
    CHECK_FOR_CUDA_ERROR();
    Point *dPoints;
    hipMalloc((void**)&dPoints, sizeof(Point) *size);
    CHECK_FOR_CUDA_ERROR();
    hipMemcpy(dPoints, data, sizeof(Point) *size, hipMemcpyHostToDevice); 
    CHECK_FOR_CUDA_ERROR();
    free(data);
    this->data = dPoints;
    
    float* dMass;
    hipMalloc((void**)&dMass, sizeof(float) * size);
    CHECK_FOR_CUDA_ERROR();
    hipMemcpy(dMass, mass, sizeof(float) * size, hipMemcpyHostToDevice);
    CHECK_FOR_CUDA_ERROR();
    free(mass);
    mass = dMass;

	hipMalloc((void**)&(pointForces), maxNumForces * sizeof(float4) * size);
    CHECK_FOR_CUDA_ERROR();
	hipMemset(pointForces, 0, sizeof(float4) * maxNumForces * size);
    CHECK_FOR_CUDA_ERROR();
}

void VertexPool::DeAlloc() {
    CHECK_FOR_CUDA_ERROR();
    hipFree(data);
    hipFree(ABC);
    hipFree(Ui_t);
    hipFree(Ui_tminusdt);
    hipFree(externalForces);
    hipFree(mass);
    hipFree(pointForces);
    CHECK_FOR_CUDA_ERROR();
}

Body::Body(unsigned int size) {
    CHECK_FOR_CUDA_ERROR();
    numTetrahedra = size;
    
    tetrahedra = 
        (Tetrahedron*) malloc(numTetrahedra*sizeof(Tetrahedron));

    hipMalloc((void**)&(shape_function_deriv), 
               sizeof(ShapeFunctionDerivatives) * numTetrahedra);

    writeIndices = (int4*)malloc(sizeof(int4) * size);
    volume = (float*)malloc(sizeof(float) * size);
    CHECK_FOR_CUDA_ERROR();
}

void Body::ConvertToCuda() {
    CHECK_FOR_CUDA_ERROR();
        Tetrahedron *dTets;
        hipMalloc((void**)&dTets, sizeof(Tetrahedron)*numTetrahedra);
        CHECK_FOR_CUDA_ERROR();
        hipMemcpy(dTets, tetrahedra,
                   sizeof(Tetrahedron)*numTetrahedra , hipMemcpyHostToDevice); 
        CHECK_FOR_CUDA_ERROR();
        free(tetrahedra);
        this->tetrahedra = dTets;

        float* dVolume;
        hipMalloc((void**)&dVolume,
                   sizeof(float) * numTetrahedra);
        CHECK_FOR_CUDA_ERROR();
        hipMemcpy(dVolume, volume,
                   sizeof(float) * numTetrahedra, hipMemcpyHostToDevice);
        CHECK_FOR_CUDA_ERROR();
        free(volume);
        this->volume = dVolume;

        int4* dWriteIndices;
        hipMalloc((void**)&(dWriteIndices),
                   sizeof(int4) * numWriteIndices);
        CHECK_FOR_CUDA_ERROR();
        hipMemcpy(dWriteIndices, writeIndices, 
                   sizeof(int4) * numWriteIndices,
                   hipMemcpyHostToDevice);
        free(writeIndices);
        writeIndices = dWriteIndices;
        CHECK_FOR_CUDA_ERROR();
}

void Body::DeAlloc() {
    CHECK_FOR_CUDA_ERROR();
    hipFree(tetrahedra);
    hipFree(shape_function_deriv);
    hipFree(writeIndices);
    hipFree(volume);
    CHECK_FOR_CUDA_ERROR();
}

void Surface::ConvertToCuda() {
    CHECK_FOR_CUDA_ERROR();
    Triangle* dTriangles;
	hipMalloc((void**)&dTriangles, sizeof(Triangle) *numFaces);
	hipMemcpy(dTriangles, faces, 
               sizeof(Triangle) *numFaces, hipMemcpyHostToDevice);
	free(faces);
    this->faces = dTriangles;
    CHECK_FOR_CUDA_ERROR();
}

void Surface::DeAlloc() {
    CHECK_FOR_CUDA_ERROR();
    hipFree(faces);
    CHECK_FOR_CUDA_ERROR();
}
