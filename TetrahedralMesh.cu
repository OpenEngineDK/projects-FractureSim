#include "hip/hip_runtime.h"
#include "TetrahedralMesh.h"
#include "CUDA.h"

VertexPool::VertexPool(unsigned int size) {
    this->size = size;
    data = (Point*) malloc(size*sizeof(Point));

	hipMalloc((void**)&(ABC), sizeof(float4) * size);
	hipMalloc((void**)&(Ui_t), sizeof(float4) * size);
	hipMalloc((void**)&(Ui_tminusdt), sizeof(float4) * size);
	hipMalloc((void**)&(externalForces), sizeof(float4) * size);
    CHECK_FOR_CUDA_ERROR();

	hipMemset(externalForces, 0, sizeof(float4) * size);
	hipMemset(Ui_t, 0, sizeof(float4) * size);
	hipMemset(Ui_tminusdt, 0, sizeof(float4) * size);
    CHECK_FOR_CUDA_ERROR();
}

void VertexPool::ConvertToCuda() {
    Point *dPoints; 
    hipMalloc((void**)&dPoints, sizeof(Point) *size);
    hipMemcpy(dPoints, data, sizeof(Point) *size, hipMemcpyHostToDevice); 
    free(data);
    this->data = dPoints;
    CHECK_FOR_CUDA_ERROR();
}

void VertexPool::DeAlloc() {
    hipFree(data);
    hipFree(ABC);
    hipFree(Ui_t);
    hipFree(Ui_tminusdt);
    hipFree(externalForces);
    hipFree(mass);
    hipFree(pointForces);
}

Body::Body(unsigned int size) {
    numTetrahedra = size;
    
    tetrahedra = 
        (Tetrahedron*) malloc(numTetrahedra*sizeof(Tetrahedron));

    hipMalloc((void**)&(shape_function_deriv), 
               sizeof(ShapeFunctionDerivatives) * numTetrahedra);
}

void Body::ConvertToCuda() {
        Tetrahedron *dTets;
        hipMalloc((void**)&dTets, sizeof(Tetrahedron)*numTetrahedra);
        hipMemcpy(dTets, tetrahedra,
                   sizeof(Tetrahedron)*numTetrahedra , hipMemcpyHostToDevice); 
        free(tetrahedra);
        this->tetrahedra = dTets;
}

void Body::DeAlloc() {
    hipFree(tetrahedra);
    hipFree(shape_function_deriv);
    hipFree(writeIndices);
    hipFree(volume);
}

void Surface::ConvertToCuda() {
    Triangle* dTriangles;
	hipMalloc((void**)&dTriangles, sizeof(Triangle) *numFaces);
	hipMemcpy(dTriangles, faces, 
               sizeof(Triangle) *numFaces, hipMemcpyHostToDevice);
	free(faces);
    this->faces = dTriangles;
}

void Surface::DeAlloc() {
    hipFree(faces);
}
